#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <time.h>
#define N 1024

// declare the kernel
__global__ void daxpy(int n, double a, double *x, double *y){
int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < N){
		y[i] += a*x[i];
	}
}

int main(void){
	double *x, *y, a, *dx, *dy;
	x = (double *)malloc(sizeof(double)*N);
	y = (double *)malloc(sizeof(double)*N);
	// initialize x and y
	srand(time(NULL));

	// allocate device memory for x and y
	hipMalloc(dx, N*sizeof(double));
	hipMalloc(dy, N*sizeof(double));
	// copy host memory to device memory
	hipMemcpy(dx, x, N*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(dy, y, N*sizeof(double), hipMemcpyHostToDevice);
	// launch the kernel function
	a=0.1;
	daxpy<<<N/64,64>>>(N, a, dx, dy);
	// copy device memory to host memory
	hipMemcpy(y, dy, N*sizeof(double), hipMemcpyDeviceToHost);
	// deallocate device memory
	cudaMemFree(dx);
	cudaMemFree(dy);
	free(x);
	free(y);
}
