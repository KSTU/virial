#include <stdio.h>
#include "messages.h"
#include "input.h"
#include "simulation.h"


int main(int argc, char *argv[]){
	int DeviceCount;
	hipDeviceProp_t dp;
	simulation sp;
	molecula substance;
	topology top;
	int i;
	//
	
	//
	hipGetDeviceCount(&DeviceCount);
	printf("Found %d device \n",DeviceCount);
	for (int device =0; device<DeviceCount;device++){
		hipGetDeviceProperties(&dp,device);
		printf("Clock rate : %d \n", dp.clockRate);
		printf("Max thread dimention %d %d %d \n", dp.maxThreadsDim[0],dp.maxThreadsDim[1],dp.maxThreadsDim[2]);
	}
	if(argc<3){
		f_usage();
		return 0;
	}
	if(check_flag(argc,argv,&sp)!=0){
		f_error("checking program parameters");
		return 1;
	}
	//read initial gro file
	if(read_gro(sp.substance_file_name,&substance)!=0){
		f_error("reading gro file");
		return 1;
	}
	//read topology
	if(read_top(sp.substance_top_name,&top)!=0){
		f_error("reading topology file");
		return 1;
	}
	//get atoms ID
	if(get_atom_id(&substance,&top)!=0){
		f_error("getting atom ID");
		return 1;
	}
	if(strcmp(sp.type,"IL")==0){
		f_message("compute for ionic liquid type");
		if (prop_boundary(&sub,&top)!=0){
			f_error("boundary fail");
			return 1;
		}
		
	}
}
