#include "hip/hip_runtime.h"
#include <stdio.h>
#include "input.h"
#include "messages.h"
#include "simulation.h"
#include "integrate.h"

int prop_boundary(molecula *sub,topology *top){
	FILE *temp_file;
	float r_cur;
	float r_min;
	float r_delta;
	int dirrection; //1 -- up, -1 -- down
	float mix_sigma;
	float mix_epsilon;
	float mix_q;
	int iter;
	float f1,f2;

	r_cur=abs(sub->x[1]-sub->x[0]);
	dirrection=1;
	r_delta=r_cur/1000;
	
	//get potential minimum
	mix_epsilon=0.1;
	mix_sigma=0.1;
	mix_q=0.1;
	while(iter<100){
		f1=potential(mix_sigma,mix_epsilon,mix_q,r_cur);
		f2=potential(mix_sigma,mix_epsilon,mix_q,r_cur+dirrection*r_delta);
		if(f1>f2){
			dirrection=-dirrection;
			r_delta=r_delta/2.0;
		}
		else{
			r_cur=r_cur+dirrection*r_delta;
		}
	}
	//get probability distribution
	
	
	//write to file
	temp_file=fopen("prop.out","w");
	
	fopen(temp_file);
	return 0;
}
