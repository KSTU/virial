#include "hip/hip_runtime.h"
#include <stdio.h>
#include "input.h"
#include "messages.h"
#include "simulation.h"
#include "integrate.h"

int prop_boundary(molecula *sub,topology *top){
	FILE *temp_file;
	float r_cur;
	float r_min;
	float r_delta;
	int dirrection; //1 -- up, -1 -- down
	float mix_sigma;
	float mix_epsilon;
	float mix_q;
	int iter;
	float f1,f2;

	r_cur=abs(sub->x[1]-sub->x[0]);
	dirrection=1;
	r_delta=r_cur/1000;
	
	//get potential minimum
	mix_epsilon=mix_geom(top->sigma[sub->atom_id[0]],top->sigma[sub->atom_id[1]]);
	mix_sigma=mix_ariph(top->epsilon[sub->atom_id[0]],top->epsilon[sub->atom_id[1]]);
	mix_q=mix_charge(top->q[sub->atom_id[0]],top->q[sub->atom_id[1]]);
	iter=0;
	while(iter<100){
		f1=potential(mix_sigma,mix_epsilon,mix_q,r_cur);
		f2=potential(mix_sigma,mix_epsilon,mix_q,r_cur+dirrection*r_delta);
		if(f1>f2){
			dirrection=-dirrection;
			r_delta=r_delta/2.0;
		}
		else{
			r_cur=r_cur+dirrection*r_delta;
		}
	}
	//get probability distribution
	
	
	//write to file
	temp_file=fopen("prop.out","w");
	
	fclose(temp_file);
	return 0;
}

float potential(float sig,float eps, float q, float r){
	float p;
	float sr;
	const float k=1.08;	//
	
	sr=sig/r;
	sr=sr*sr;	//2
	sr=sr*sr*sr;	//6
	p=4.0*eps*(sr*sr-sr)+k*q/r;
	
	return 0.2;
}

float mix_ariph(float s1,float s2){
	return (s1+s2)/2.0;
}

float mix_geom(float s1, float s2){
	return sqrt(s1*s2);
}
float mix_charge(float q1,float q2){
	return q1*q2;
}
